#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int getnextnum(FILE* f, int* val){
        char num[6];
        int idx = 0;
        char c;
        int ret = 0;
        while(1) {
                num[idx] = '\0';
                c = getc(f);
                if(c == EOF) {
                        ret = 1;
                        break;
                }
                if(c == ',') {
                        c = getc(f);
                        break;
                }
                num[idx] = c;
                idx++;
        }
        *val = atoi(num);
        return ret;
}

int* getarr(int* arrlen) {
        FILE* inp = fopen("./inp.txt", "r");
        int val;
        int count = 0;
        int len = 0;
        int* arr = (int*)malloc(1 * sizeof(int));
        int* transfer;
        int end = 0;
        while(!end) {
                if(count == len) {
                        len += 10;
                        transfer = (int*)malloc(len * sizeof(int));
                        memcpy(transfer, arr, count * sizeof(int));
                        free(arr);
                        arr = transfer;
                }
                end = getnextnum(inp, &val);
                arr[count] = val;
                count++;
        }
        fclose(inp);
        transfer = (int*)malloc(count * sizeof(int));
        memcpy(transfer, arr, count * sizeof(int));
	free(arr);
	arr = transfer;
	*arrlen = count;
	return arr;
}
