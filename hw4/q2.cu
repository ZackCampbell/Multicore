#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <sm_35_atomic_functions.h>

int* partA(int * A, int * B, int count);
int* partB(int * A, int * B, int count);
int* partC(int * B, int * C, int count);

#define OUTPUT_SIZE 10

int getnextnum(FILE* f, int* val) {
	char num[6];
	int idx = 0;
	char c;
	int ret = 0;
	while (1) {
		num[idx] = '\0';
		c = getc(f);
		if (c == EOF) {
			ret = 1;
			break;
		}
		if (c == ',') {
			c = getc(f);
			break;
		}
		num[idx] = c;
		idx++;
	}
	*val = atoi(num);
	return ret;
}

int* getarr(int* arrlen) {
	FILE* inp = fopen("../MulticoreHW4CUDA/inp.txt", "r");
	int val;
	int count = 0;
	int len = 0;
	int* arr = (int*)malloc(1 * sizeof(int));
	int* transfer;
	int end = 0;
	while (!end) {
		if (count == len) {
			len += 10;
			transfer = (int*)malloc(len * sizeof(int));
			memcpy(transfer, arr, count * sizeof(int));
			free(arr);
			arr = transfer;
		}
		end = getnextnum(inp, &val);
		arr[count] = val;
		count++;
	}
	fclose(inp);
	transfer = (int*)malloc(count * sizeof(int));
	memcpy(transfer, arr, count * sizeof(int));
	free(arr);
	arr = transfer;
	*arrlen = count;
	return arr;
}

__global__ void partAKernel(int* A, int* B, int len) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	for (int i = index; i < len; i += blockDim.x * gridDim.x) {
		atomicAdd(&B[A[i]/100], 1);
	}
}

__global__ void partBKernel(int* A, int* B, int len) {
	__shared__ int localA[OUTPUT_SIZE];
	__shared__ int localB[OUTPUT_SIZE];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x;
	localA[lindex] = A[gindex] / 100;
	localB[lindex] = 0;
	__syncthreads();

	if (gindex < len) {
		atomicAdd(&localB[localA[lindex]], 1);
	}
	
	__syncthreads();

	atomicAdd(&B[lindex], localB[lindex]);
}

__global__ void partCKernel(int* B, int* C, int len) {
	int thid = threadIdx.x;
	C[thid] = B[thid];
	__syncthreads();
	for (int offset = 1; offset < len; offset *= 2) {
		if (thid - offset >= 0)
			C[thid] += C[thid - offset];
		__syncthreads();
	}
}

int main() {
	int count;
	int* A = getarr(&count);
	
	int* partAOutput = (int*)malloc(sizeof(int) * 10);
	int* partBOutput = (int*)malloc(sizeof(int) * 10);
	int* partCOutput = (int*)malloc(sizeof(int) * 16);
	for (int i = 0; i < 10; i++) {
		partAOutput[i] = 0;
		partBOutput[i] = 0;
	}
	for (int i = 0; i < 16; i++) {
		partCOutput[i] = 0;
	}

	int* partAAnswer = partA(A, partAOutput, count);
	int* partBAnswer = partB(A, partBOutput, count);
	int* partCAnswer = partC(partAOutput, partCOutput, 16);
	
	FILE * aOut = fopen("q2a.txt", "w");
	FILE * bOut = fopen("q2b.txt", "w");
	FILE * cOut = fopen("q2c.txt", "w");
	for (int i = 0; i < 10; i++) {
		fprintf(aOut, "%d, ", partAAnswer[i]);
		fprintf(bOut, "%d, ", partBAnswer[i]);
		fprintf(cOut, "%d, ", partCAnswer[i]);
	}
	fclose(aOut);
	fclose(bOut);
	fclose(cOut);

	free(partAOutput);
	free(partBOutput);
	free(partCOutput);
	free(A);
	free(partAAnswer);
	free(partBAnswer);
	free(partCAnswer);
	return 0;
}

int* partA(int* A, int* B, int count) {
	int* d_A;
	int* d_B;

	hipMalloc((void**)&d_A, count * sizeof(int));
	hipMalloc((void**)&d_B, OUTPUT_SIZE * sizeof(int));

	hipMemcpy(d_A, A, count * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid((count / 2) + (count % 2), 1);
	dim3 dimBlock(1, 1, 1);

	partAKernel<<< dimGrid, dimBlock >>>(d_A, d_B, count);

	hipDeviceSynchronize();
	hipMemcpy(B, d_B, OUTPUT_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_B);
	hipFree(d_A);

	int* result = (int*)malloc(sizeof(int) * 10);
	memcpy(result, B, OUTPUT_SIZE * sizeof(int));
	return result;
}

int* partB(int* A, int* B, int count) {
	int* d_A;
	int* d_B;

	hipMalloc((void**)&d_A, count * sizeof(int));
	hipMalloc((void**)&d_B, OUTPUT_SIZE * sizeof(int));

	dim3 dimGrid(1024, 1);
	dim3 dimBlock(10, 1, 1);

	hipMemcpy(d_A, A, count * sizeof(int), hipMemcpyHostToDevice);

	partBKernel <<< dimGrid, dimBlock >>> (d_A, d_B, count);

	hipDeviceSynchronize();
	hipMemcpy(B, d_B, OUTPUT_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_B);
	hipFree(d_A);

	int* result = (int*)malloc(sizeof(int) * 10);
	memcpy(result, B, OUTPUT_SIZE * sizeof(int));
	return result;
}

int* partC(int* B, int* C, int count) {
	int* d_B;
	int* d_C;

	int* paddedB = (int*)malloc(sizeof(int) * 16);
	for (int i = 0; i < 16; i++) {
		paddedB[i] = 0;
	}
	for (int i = 0; i < 10; i++) {
		paddedB[i + 6] = B[i];
	}

	hipMalloc((void**)&d_B, 16 * sizeof(int));
	hipMalloc((void**)&d_C, 16 * sizeof(int));

	dim3 dimGrid(1, 1);
	dim3 dimBlock(16, 1, 1);

	hipMemcpy(d_B, paddedB, 16 * sizeof(int), hipMemcpyHostToDevice);

	partCKernel <<< dimGrid, dimBlock >>> (d_B, d_C, 16);

	hipDeviceSynchronize();
	hipMemcpy(C, d_C, 16 * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_C);
	hipFree(d_B);

	free(paddedB);

	int* output = (int*)malloc(sizeof(int) * 10);
	for (int i = 0; i < 10; i++) {
		output[i] = C[i + 6];
	}

	int* result = (int*)malloc(sizeof(int) * 10);
	memcpy(result, output, 10 * sizeof(int));
	free(output);
	return result;
}