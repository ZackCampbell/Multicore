#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "parse.h"

__global__ void comparevals(int* src, int* dest, int len) {
	int s1, s2, d, val1, val2, zerolen;
	zerolen = len - 1;
	d = blockIdx.x;
	s1 = d * 2;
	s2 = s1 + 1;
	if (s1 + 1 > zerolen) {
		return;
	}
	val1 = src[s1];
	if (s2 + 1 > zerolen) {
		dest[d] = val1;
		return;
	}
	val2 = src[s2];
	dest[d] = ((val1 < val2) ? val1 : val2);
}

int main(int argc, char** argv) {
	bool DEBUG = true;
	int count;
	int* arr = getarr(&count);
        for(int i = 0; i < count; i++){
                printf("%d, ", arr[i]);
        }
	printf("\n\n");
	int* csrc; 
	int* cdest;
	hipMalloc(&csrc, count * sizeof(int));
	hipMalloc(&cdest, count * sizeof(int));
	hipMemcpy(arr, csrc, count, hipMemcpyHostToDevice);
	dim3 dimGrid((count / 2) + (count % 2), 1);
	dim3 dimBlock(1, 1, 1);	
	for(int i = count; i == 0; i = (i / 2) + (i % 2)) {
		comparevals<<<dimGrid, dimBlock>>>(csrc, cdest, i);
		hipMemcpy(cdest, csrc, count, hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();
	}
	hipMemcpy(cdest, arr, count, hipMemcpyDeviceToHost);
	hipFree(cdest);
	hipFree(csrc);
	printf("%d, ", arr[0]);
	free(arr);
	return 0;
}

