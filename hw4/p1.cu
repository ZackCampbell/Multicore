#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "parse.h"

int getnextnum(FILE* f, int* val){
        char num[6];
        int idx = 0;
        char c;
        int ret = 0;
        while(1) {
                num[idx] = '\0';
                c = getc(f);
                if(c == EOF) {
                        ret = 1;
                        break;
                }
                if(c == ',') {
                        c = getc(f);
                        break;
                }
                num[idx] = c;
                idx++;
        }
        *val = atoi(num);
        return ret;
}

int* getarr(int* arrlen) {
        FILE* inp = fopen("./inp.txt", "r");
        int val;
        int count = 0;
        int len = 0;
        int* arr = (int*)malloc(1 * sizeof(int));
        int* transfer;
        int end = 0;
        while(!end) {
                if(count == len) {
                        len += 10;
                        transfer = (int*)malloc(len * sizeof(int));
                        memcpy(transfer, arr, count * sizeof(int));
                        free(arr);
                        arr = transfer;
                }
                end = getnextnum(inp, &val);
                arr[count] = val;
                count++;
        }
        fclose(inp);
        transfer = (int*)malloc(count * sizeof(int));
        memcpy(transfer, arr, count * sizeof(int));
        free(arr);
        arr = transfer;
        *arrlen = count;
        return arr;
}

__global__ void comparevals(int* src, int* dest, int len) {
	int s1, s2, d, val1, val2, zerolen;
	zerolen = len - 1;
	d = blockIdx.x;
	s1 = d * 2;
	s2 = s1 + 1;
	if (s1 > zerolen) {
		return;
	}
	val1 = src[s1];
	if (s2 > zerolen) {
		dest[d] = val1;
		return;
	}
	val2 = src[s2];
	dest[d] = ((val1 < val2) ? val1 : val2);
}

int getmin() {
	int count, min;
	int* arr = getarr(&count);
	int* csrc; 
	int* cdest;
	hipMalloc((void**) &csrc, count * sizeof(int));
	hipMalloc((void**) &cdest, count * sizeof(int));
	hipMemcpy((void*) csrc, (void*) arr, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy((void*) cdest, (void*) arr, count * sizeof(int), hipMemcpyHostToDevice);
	for(int i = count; i != 1; i = (i / 2) + (i % 2)) {
		comparevals<<<((count/2) + (count%2)), 1>>>(csrc, cdest, i);
		hipDeviceSynchronize();
		hipMemcpy((void*) csrc, (void*) cdest, count * sizeof(int), hipMemcpyDeviceToDevice);
	}
	hipMemcpy((void*) arr, (void*) cdest, count * sizeof(int), hipMemcpyDeviceToHost);
	min = arr[0];	
	hipFree(cdest);
	hipFree(csrc);
	free(arr);
	return min;
}

__global__ void lastdigit(int* arr) {
	arr[blockIdx.x] = arr[blockIdx.x] % 10;
}

int* getlastdigits(int* len) {
	int* arr = getarr(len);
	int mylen = *len;
	int* carr;
	hipMalloc((void**) &carr, mylen * sizeof(int));
	hipMemcpy((void*) carr, (void*) arr, mylen * sizeof(int), hipMemcpyHostToDevice);
	lastdigit<<<mylen, 1>>>(carr);
	hipDeviceSynchronize();
	hipMemcpy((void*) arr, (void*) carr, mylen * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(carr);
	return arr;
}

int main(int argc, char** argv) {
	int min;
	min = getmin();
	FILE* f = fopen("./q1a.txt", "w");
	fprintf(f, "%d", min);
	fclose(f);
	
	int len;	
	int* ldarr = getlastdigits(&len);
	f = fopen("./q1b.txt", "w");
	for(int i = 0; i < len; i++) {
		if(i == len - 1) fprintf(f, "%d", ldarr[i]);	
		else fprintf(f, "%d, ", ldarr[i]);
	}
	free(ldarr);	
	return 0;
}

