#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void comparevals(int* src, int* dest, int len) {
	int s1, s2, d, val1, val2, zerolen;
	zerolen = len - 1;
	d = blockIdx.x;
	s1 = d * 2;
	s2 = s1 + 1;
	if (s1 + 1 > zerolen) {
		return;
	}
	val1 = src[s1];
	if (s2 + 1 > zerolen) {
		dest[d] = val1;
		return;
	}
	val2 = src[s2];
	dest[d] = ((val1 < val2) ? val1 : val2);
}



int main(int argc, char** argv) {
	FILE* inp = fopen("./inp.txt", "r");
	
