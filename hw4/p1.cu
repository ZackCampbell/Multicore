#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "parse.h"

__global__ void comparevals(int* src, int* dest, int len) {
	int s1, s2, d, val1, val2, zerolen;
	zerolen = len - 1;
	d = blockIdx.x;
	s1 = d * 2;
	s2 = s1 + 1;
	if (s1 + 1 > zerolen) {
		return;
	}
	val1 = src[s1];
	if (s2 + 1 > zerolen) {
		dest[d] = val1;
		return;
	}
	val2 = src[s2];
	dest[d] = ((val1 < val2) ? val1 : val2);
}

int getmin() {
	int count, min;
	int* arr = getarr(&count);
	int* csrc; 
	int* cdest;
	hipMalloc((void**) &csrc, count * sizeof(int));
	hipMalloc((void**) &cdest, count * sizeof(int));
	hipMemcpy((void*) csrc, (void*) arr, count * sizeof(int), hipMemcpyHostToDevice);
	for(int i = count; i != 1; i = (i / 2) + (i % 2)) {
		comparevals<<<((count/2) + (count%2)), 1>>>(csrc, cdest, i);
		hipDeviceSynchronize();
		hipMemcpy((void*) csrc, (void*) cdest, count * sizeof(int), hipMemcpyDeviceToDevice);
	}
	hipMemcpy((void*) arr, (void*) cdest, count * sizeof(int), hipMemcpyDeviceToHost);
	min = arr[0];	
	hipFree(cdest);
	hipFree(csrc);
	free(arr);
	return min;
}

__global__ void lastdigit(int* arr) {
	arr[blockIdx.x] = arr[blockIdx.x] % 10;
}

int* getlastdigits(int* len) {
	int* arr = getarr(len);
	int mylen = *len;
	int* carr;
	hipMalloc((void**) &carr, mylen * sizeof(int));
	hipMemcpy((void*) carr, (void*) arr, mylen * sizeof(int), hipMemcpyHostToDevice);
	lastdigit<<<mylen, 1>>>(carr);
	hipDeviceSynchronize();
	hipMemcpy((void*) arr, (void*) carr, mylen * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(carr);
	return arr;
}

int main(int argc, char** argv) {
	int min;
	min = getmin();
	FILE* f = fopen("./q1a.txt", "w");
	fprintf(f, "%d", min);
	fclose(f);
	
	int len;	
	int* ldarr = getlastdigits(&len);
	f = fopen("./q1b.txt", "w");
	for(int i = 0; i < len; i++) {
		if(i == len - 1) fprintf(f, "%d", ldarr[i]);	
		else fprintf(f, "%d, ", ldarr[i]);
	}
	free(ldarr);	
	return 0;
}

