#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "parse.h"
#include <sm_35_atomic_functions.h>

int* partA(int * A, int * B, int count);
int* partB(int * A, int * B, int count);

#define OUTPUT_SIZE 10

__global__ void partAKernel(int* A, int* B, int len) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	for (int i = index; i < len; i += blockDim.x * gridDim.x) {
		atomicAdd(&B[A[i]/100], 1);
	}
}

__global__ void partBKernel(int* A, int* B, int len) {
	__shared__ int localA[OUTPUT_SIZE];
	__shared__ int localB[OUTPUT_SIZE];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x;
	localA[lindex] = A[gindex] / 100;
	localB[lindex] = 0;
	__syncthreads();

	if (gindex < len) {
		atomicAdd(&localB[localA[lindex]], 1);
	}
	
	__syncthreads();

	atomicAdd(&B[lindex], localB[lindex]);
}

int main() {
	int count;
	int* A = getarr(&count);
	
	int* partAOutput = (int*)malloc(sizeof(int) * 10);
	for (int i = 0; i < 10; i++) {
		partAOutput[i] = 0;
	}
	int* partBOutput = (int*)malloc(sizeof(int) * 10);
	for (int i = 0; i < 10; i++) {
		partBOutput[i] = 0;
	}

	int* partAAnswer = partA(A, partAOutput, count);
	int* partBAnswer = partB(A, partBOutput, count);
	for (int i = 0; i < 10; i++) {
		printf("%d, ", partBAnswer[i]);
	}
	
	free(partAOutput);
	free(partBOutput);
	free(A);
	free(partAAnswer);
	free(partBAnswer);
	return 0;
}

int* partA(int* A, int* B, int count) {
	int* d_A;
	int* d_B;

	hipMalloc((void**)&d_A, count * sizeof(int));
	hipMalloc((void**)&d_B, OUTPUT_SIZE * sizeof(int));

	hipMemcpy(d_A, A, count * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid((count / 2) + (count % 2), 1);
	dim3 dimBlock(1, 1, 1);

	partAKernel<<< dimGrid, dimBlock >>>(d_A, d_B, count);

	hipDeviceSynchronize();
	hipMemcpy(B, d_B, OUTPUT_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_B);
	hipFree(d_A);

	int* result = (int*)malloc(sizeof(int) * 10);
	memcpy(result, B, OUTPUT_SIZE * sizeof(int));
	return result;
}

int* partB(int* A, int* B, int count) {
	int* d_A;
	int* d_B;

	hipMalloc((void**)&d_A, count * sizeof(int));
	hipMalloc((void**)&d_B, OUTPUT_SIZE * sizeof(int));

	dim3 dimGrid(1024, 1);
	dim3 dimBlock(10, 1, 1);

	hipMemcpy(d_A, A, count * sizeof(int), hipMemcpyHostToDevice);

	partBKernel <<< dimGrid, dimBlock >>> (d_A, d_B, count);

	hipDeviceSynchronize();
	hipMemcpy(B, d_B, OUTPUT_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_B);
	hipFree(d_A);

	int* result = (int*)malloc(sizeof(int) * 10);
	memcpy(result, B, OUTPUT_SIZE * sizeof(int));
	return result;
}