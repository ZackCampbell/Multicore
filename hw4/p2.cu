#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "parse.h"
#include <sm_35_atomic_functions.h>

int* partA(int * A, int * B, int count);

__global__ void partAKernel(int* A, int* B, int len) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	for (int i = index; i < len; i += blockDim.x * gridDim.x) {
		atomicAdd(&B[A[i]/100], 1);
	}
}


int main() {
	int count;
	int* A = getarr(&count);
	
	int* B = (int*)malloc(sizeof(int) * 10);
	for (int i = 0; i < 10; i++) {
		B[i] = 0;
	}

	int* partAAnswer = partA(A, B, count);
	
	free(B);
	free(A);
	free(partAAnswer);
	return 0;
}

int* partA(int* A, int* B, int count) {
	int* d_A;
	int* d_B;

	hipMalloc((void**)&d_A, count * sizeof(int));
	hipMalloc((void**)&d_B, 10 * sizeof(int));

	hipMemcpy(d_A, A, count * sizeof(int), hipMemcpyHostToDevice);

	partAKernel<<< 128, 128 >>>(d_A, d_B, count);

	hipDeviceSynchronize();
	hipMemcpy(B, d_B, 10 * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_B);
	hipFree(d_A);

	int* result = (int*)malloc(sizeof(int) * 10);
	memcpy(result, B, 10 * sizeof(int));
	return result;
}