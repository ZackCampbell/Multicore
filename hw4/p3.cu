#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "parse.h"

__global__ markodds(int* src, int* dest) {
	if(src[blockIdx.x] % 2 == 0) dest[blockIdx.x] = 0;
	else dest[blockIdx.x] = 1;
}

__global__ prefixsum(int* src, int* dest) {
	

int main(int argc, char** argv) {
	int len;
	int* arr = getarr(&len);

	int* carr; int* cisodd; int* cpresum;
	hipMalloc((void**) &carr, len*sizeof(int));
	hipMalloc((void**) &cisodd, len*sizeof(int));
	hipMalloc((void**) &cpresum, len*sizeof(int));
	hipMemcpy((void*) carr, (void*) arr, len*sizeof(int));

	markodds<<<len, 1>>>(carr, cisodd);
	markodds<<<len, 1>>>(cisodd, cpresum);

	hipFree(carr);
	hipFree(cisodd);
	hipFree(cpresum);
	free(arr);
	free(isodd);
	return 0;
}

