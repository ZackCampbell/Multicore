#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "parse.h"

int getnextnum(FILE* f, int* val){
        char num[6];
        int idx = 0;
        char c;
        int ret = 0;
        while(1) {
                num[idx] = '\0';
                c = getc(f);
                if(c == EOF) {
                        ret = 1;
                        break;
                }
                if(c == ',') {
                        c = getc(f);
                        break;
                }
                num[idx] = c;
                idx++;
        }
        *val = atoi(num);
        return ret;
}

int* getarr(int* arrlen) {
        FILE* inp = fopen("./inp.txt", "r");
        int val;
        int count = 0;
        int len = 0;
        int* arr = (int*)malloc(1 * sizeof(int));
        int* transfer;
        int end = 0;
        while(!end) {
                if(count == len) {
                        len += 10;
                        transfer = (int*)malloc(len * sizeof(int));
                        memcpy(transfer, arr, count * sizeof(int));
                        free(arr);
                        arr = transfer;
                }
                end = getnextnum(inp, &val);
                arr[count] = val;
                count++;
        }
        fclose(inp);
        transfer = (int*)malloc(count * sizeof(int));
        memcpy(transfer, arr, count * sizeof(int));
        free(arr);
        arr = transfer;
        *arrlen = count;
        return arr;
}

__global__ void markodds(int* src, int* dest) {
	if(src[blockIdx.x] % 2 == 0) dest[blockIdx.x] = 0;
	else dest[blockIdx.x] = 1;
}

__global__ void presum(int* src, int* dest, int len, int offset, int i) {
	int id = blockIdx.x;
	int idx1, idx2;
	if(id < i) {
		idx1 = offset * (2 * id + 1) - 1;
		idx2 = offset * (2 * id + 2) - 1;
		dest[idx2] += dest[idx1];
	}
}

__global__ void midsum(int* dest, int len) {
	int id = blockIdx.x;
	if(id == 0) dest[len - 1] = 0;
}

__global__ void postsum(int* src, int* dest, int len, int offset, int i){
	int hold, idx1, idx2;
	int id = blockIdx.x;
	if(id < i) {
		idx1 = offset * (2 * id + 1) - 1;
		idx2 = offset * (2 * id + 2) - 1;
		hold = dest[idx1];
		dest[idx1] = dest[idx2];
		dest[idx2] += hold;
	}
}

__global__ void genresult(int* src1, int* src2, int* src3, int* dest) {
	if(src2[blockIdx.x] != 1) return;	
	int idx = src3[blockIdx.x];
	int val = src1[blockIdx.x];
	dest[idx] = val;
}

int main(int argc, char** argv) {
	int len;
	int* arr = getarr(&len);

	// zero extend arr to first largest power of 2
	for(int i = 2; ; i *= 2) {
		if(i > len) {
			int* tmp = (int*) malloc(i * sizeof(int));
			for(int j = 0; j < i; j++) {
				if(j < len) tmp[j] = arr[j];
				else tmp[j] = 0;
			}
			len = i;
			free(arr);
			arr = tmp;
			break;
		}	
	}

	int* carr; int* cisodd; int* cpresum; int* cresult;
	hipMalloc((void**) &carr, len*sizeof(int));
	hipMalloc((void**) &cisodd, len*sizeof(int));
	hipMalloc((void**) &cpresum, len*sizeof(int));
	hipMalloc((void**) &cresult, len*sizeof(int));
	hipMemcpy((void*) carr, (void*) arr, len*sizeof(int), hipMemcpyHostToDevice);

	markodds<<<len, 1>>>(carr, cisodd);
	hipDeviceSynchronize();

	hipMemcpy((void*) cpresum, (void*) cisodd, len*sizeof(int), hipMemcpyDeviceToDevice);
	int offset = 1;
	for(int i = len >> 1; i > 0; i = i >> 1) {
		hipDeviceSynchronize();
		presum<<<len, 1>>>(cisodd, cpresum, len, offset, i);
		hipDeviceSynchronize();
		offset *= 2;
	}

	midsum<<<len, 1>>>(cpresum, len);	
	hipDeviceSynchronize();

	for(int i = 1; i < len; i *= 2) {
		offset >>= 1;
		hipDeviceSynchronize();
		postsum<<<len, 1>>>(cisodd, cpresum, len, offset, i);	
		hipDeviceSynchronize();
	}

	genresult<<<len, 1>>>(carr, cisodd, cpresum, cresult);
	hipDeviceSynchronize();
	
	int reslen;
	hipMemcpy((void*) &reslen, (void*) &cpresum[len - 1], sizeof(int), hipMemcpyDeviceToHost);
	reslen += arr[len - 1];
	int* result = (int*) malloc(reslen * sizeof(int));
	hipMemcpy((void*) result, (void*) cresult, reslen * sizeof(int), hipMemcpyDeviceToHost);

	FILE* f = fopen("./q3.txt", "w");
	for(int i = 0; i < reslen; i++) {
		if(i == reslen - 1) fprintf(f, "%d", result[i]);
		else fprintf(f, "%d, ", result[i]);
	}

	hipFree(carr);
	hipFree(cisodd);
	hipFree(cpresum);
	hipFree(cresult);
	free(arr);
	free(result);
	return 0;
}

